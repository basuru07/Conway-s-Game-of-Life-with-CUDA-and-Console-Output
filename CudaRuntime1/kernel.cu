#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <chrono>
#include <thread>

const int WINDOW_WIDTH = 80;  // Reduced width for console output
const int WINDOW_HEIGHT = 40;  // Reduced height for console output
const int CELL_SIZE = 1;  // No need for cell size, just 1x1 grid in console
const int GRID_WIDTH = WINDOW_WIDTH / CELL_SIZE;
const int GRID_HEIGHT = WINDOW_HEIGHT / CELL_SIZE;
const int FRAME_DELAY = 100;

__global__ void updateGrid(bool* d_grid, bool* d_newGrid) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= GRID_WIDTH || y >= GRID_HEIGHT) return;

    int aliveNeighbors = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (dx == 0 && dy == 0) continue;
            int nx = (x + dx + GRID_WIDTH) % GRID_WIDTH;
            int ny = (y + dy + GRID_HEIGHT) % GRID_HEIGHT;
            aliveNeighbors += d_grid[ny * GRID_WIDTH + nx];
        }
    }

    bool currentCell = d_grid[y * GRID_WIDTH + x];
    bool newState = (currentCell && (aliveNeighbors == 2 || aliveNeighbors == 3)) ||
        (!currentCell && aliveNeighbors == 3);
    d_newGrid[y * GRID_WIDTH + x] = newState;
}

void initializeGrid(bool* grid) {
    for (int i = 0; i < GRID_WIDTH * GRID_HEIGHT; i++) {
        grid[i] = rand() % 2;
    }
}

void printGrid(bool* grid) {
    for (int y = 0; y < GRID_HEIGHT; y++) {
        for (int x = 0; x < GRID_WIDTH; x++) {
            if (grid[y * GRID_WIDTH + x]) {
                printf("O");  // Alive cell
            }
            else {
                printf(" ");  // Dead cell
            }
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char* argv[]) {
    bool* h_grid = new bool[GRID_WIDTH * GRID_HEIGHT];
    bool* h_newGrid = new bool[GRID_WIDTH * GRID_HEIGHT];
    bool* d_grid;
    bool* d_newGrid;

    initializeGrid(h_grid);

    hipMalloc(&d_grid, GRID_WIDTH * GRID_HEIGHT * sizeof(bool));
    hipMalloc(&d_newGrid, GRID_WIDTH * GRID_HEIGHT * sizeof(bool));

    hipMemcpy(d_grid, h_grid, GRID_WIDTH * GRID_HEIGHT * sizeof(bool), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((GRID_WIDTH + blockSize.x - 1) / blockSize.x, (GRID_HEIGHT + blockSize.y - 1) / blockSize.y);

    bool quit = false;
    while (!quit) {
        updateGrid << <gridSize, blockSize >> > (d_grid, d_newGrid);
        hipDeviceSynchronize();

        hipMemcpy(h_grid, d_newGrid, GRID_WIDTH * GRID_HEIGHT * sizeof(bool), hipMemcpyDeviceToHost);

        // Clear console and print grid
        system("clear");  // Use "cls" if on Windows
        printGrid(h_grid);

        std::swap(d_grid, d_newGrid);
        std::this_thread::sleep_for(std::chrono::milliseconds(FRAME_DELAY));
    }

    hipFree(d_grid);
    hipFree(d_newGrid);
    delete[] h_grid;
    delete[] h_newGrid;

    return 0;
}
